#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <agency/experimental.hpp>
#include <vector>
#include <cassert>
#include <iostream>
#include <ctime>

// allocate data in GPU memory
using Matrix = std::vector<size_t, agency::cuda::managed_allocator<size_t>>;

void sequentialMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_rows = agency::experimental::tile_evenly(right_data, n);

    agency::bulk_invoke(agency::seq(n), [=](agency::sequenced_agent& outer)
    {
        auto left_row = left_rows[outer.index()];
        agency::bulk_invoke(agency::seq(n), [=](agency::sequenced_agent& inner)
        {
            auto right_row = right_rows[inner.index()];
            for (int k = 0; k < n; ++k) {
                out_ptr[n * outer.index() + inner.index()] += left_row[k] * right_row[k];
            }
        });
    });
}

void parallelCpuMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_rows = agency::experimental::tile_evenly(right_data, n);

    agency::bulk_invoke(agency::par(n), [=](agency::parallel_agent& outer)
    {
        auto left_row = left_rows[outer.index()];
        agency::bulk_invoke(agency::par(n), [=](agency::parallel_agent& inner)
        {
            auto right_row = right_rows[inner.index()];
            for (int k = 0; k < n; ++k) {
                out_ptr[n * outer.index() + inner.index()] += left_row[k] * right_row[k];
            }
        });
    });
}

void parallelSingleGpuMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_rows = agency::experimental::tile_evenly(right_data, n);
    agency::cuda::grid_executor gpu;

    agency::bulk_invoke(agency::par(n).on(gpu), [=] __device__ (agency::parallel_agent& outer)
    {
        auto left_row = left_rows[outer.index()];
        agency::bulk_invoke(agency::par(n).on(gpu), [=] __device__ (agency::parallel_agent& inner)
        {
            auto right_row = right_rows[inner.index()];
            for (int k = 0; k < n; ++k) {
                out_ptr[n * outer.index() + inner.index()] += left_row[k] * right_row[k];
            }
        });
    });
}

void parallelAllGpuMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_rows = agency::experimental::tile_evenly(right_data, n);
    agency::cuda::multidevice_executor all_gpus;

    agency::bulk_invoke(agency::par(n).on(all_gpus), [=] __device__ (agency::parallel_agent& outer)
    {
        auto left_row = left_rows[outer.index()];
        agency::bulk_invoke(agency::par(n).on(all_gpus), [=] __device__ (agency::parallel_agent& inner)
        {
            auto right_row = right_rows[inner.index()];
            for (int k = 0; k < n; ++k) {
                out_ptr[n * outer.index() + inner.index()] += left_row[k] * right_row[k];
            }
        });
    });
}

int main()
{
    size_t n = 1 << 10;

    Matrix a(n*n, 1);
    Matrix b(n*n, 1);
    Matrix c(n*n, 0);

    Matrix reference(n*n, n);
    
    // execute sequentially in the current thread
    clock_t begin_time = clock();
    sequentialMultiply(a, b, c, n);
    clock_t difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Sequential Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on the CPU
    begin_time = clock();
    parallelCpuMultiply(a, b, c, n);
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel CPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on a GPU
    begin_time = clock();
    parallelSingleGpuMultiply(a, b, c, n);
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel single GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on all GPUs in the system
    begin_time = clock();
    parallelAllGpuMultiply(a, b, c, n);
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel All GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // Success!
    std::cout << "OK" << std::endl;
    return 0;
}
