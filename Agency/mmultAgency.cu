#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <vector>
#include <cassert>
#include <iostream>
#include <ctime>

int main()
{
    using namespace agency;
    // allocate data in GPU memory
    using matrix = std::vector<size_t, cuda::managed_allocator<size_t>>;
    
    size_t n = 1 << 10;

    matrix a(n*n, 1);
    matrix b(n*n, 1);
    matrix c(n*n, 0);

    matrix reference(n*n, n);

    size_t* a_ptr = a.data();
    size_t* b_ptr = b.data();
    size_t* c_ptr = c.data();
    
    // execute sequentially in the current thread
    clock_t begin_time = clock();
    bulk_invoke(seq(n*n), [=](sequenced_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[n*row + col] += a_ptr[n*row + k] *
                                  b_ptr[n*k + col];
        }
    });
    clock_t difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Sequential Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on the CPU
    begin_time = clock();
    bulk_invoke(par(n*n), [=](parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[n*row + col] += a_ptr[n*row + k] *
                                  b_ptr[n*k + col];
        }
    });
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel CPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on a GPU
    begin_time = clock();
    cuda::grid_executor gpu;
    bulk_invoke(par(n*n).on(gpu), [=] __device__ (parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[n*row + col] += a_ptr[n*row + k] *
                                  b_ptr[n*k + col];
        }
    });
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel Single GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on all GPUs in the system
    begin_time = clock();
    cuda::multidevice_executor all_gpus;
    bulk_invoke(par(n).on(all_gpus), [=] __device__ (parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[n*row + col] += a_ptr[n*row + k] *
                                  b_ptr[n*k + col];
        }
    });
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel All GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    std::cout << "OK" << std::endl;
    return 0;
}
