#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <agency/experimental.hpp>
#include <vector>
#include <cassert>
#include <iostream>
#include <ctime>

// allocate data in GPU memory
using Matrix = std::vector<size_t, agency::cuda::managed_allocator<size_t>>;

void sequentialMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_cols = agency::experimental::tile_evenly(right_data, n);

    agency::size2 shape{n,n};

    agency::bulk_invoke(agency::seq(shape), [=](agency::sequenced_agent_2d& self)
    {
        size_t row = self.index()[0];
        size_t col = self.index()[1];
        auto left_row = left_rows[row];
        auto right_col = right_cols[col];

        for (int k = 0; k < n; ++k) {
            out_ptr[n * row + col] += left_row[k] * right_col[k];
        }
    });
}

void parallelCpuMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_cols = agency::experimental::tile_evenly(right_data, n);

    agency::size2 shape{n,n};

    agency::bulk_invoke(agency::par(shape), [=](agency::parallel_agent_2d& self)
    {
        size_t row = self.index()[0];
        size_t col = self.index()[1];
        auto left_row = left_rows[row];
        auto right_col = right_cols[col];

        for (int k = 0; k < n; ++k) {
            out_ptr[n * row + col] += left_row[k] * right_col[k];
        }
    });
}

void parallelGpuMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_cols = agency::experimental::tile_evenly(right_data, n);

    agency::size2 shape{n,n};
    agency::cuda::parallel_executor gpu;

    agency::bulk_invoke(agency::par(shape).on(gpu), [=] __device__ (agency::parallel_agent_2d& self)
    {
        size_t row = self.index()[0];
        size_t col = self.index()[1];
        auto left_row = left_rows[row];
        auto right_col = right_cols[col];

        for (int k = 0; k < n; ++k) {
            out_ptr[n * row + col] += left_row[k] * right_col[k];
        }
    });
}

int main()
{
    size_t n = 1 << 10;

    // N.B. All multiply functions expect the right matrix to be transposed before being called.
    // However, we are only using symmetric matrices here so that doesn't matter
    Matrix a(n*n, 1);
    Matrix b(n*n, 1);
    Matrix c(n*n, 0);

    Matrix reference(n*n, n);

    // execute sequentially in the current thread
    clock_t begin_time = clock();
    sequentialMultiply(a, b, c, n);
    clock_t difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Sequential Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on the CPU
    begin_time = clock();
    parallelCpuMultiply(a, b, c, n);
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel CPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on a GPU
    begin_time = clock();
    parallelGpuMultiply(a, b, c, n);
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // Success!
    std::cout << "OK" << std::endl;
    return 0;
}
