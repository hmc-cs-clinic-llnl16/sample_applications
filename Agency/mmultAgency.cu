#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <vector>
#include <cassert>
#include <iostream>
#include <ctime>

// allocate data in GPU memory
using Matrix = std::vector<size_t, agency::cuda::managed_allocator<size_t>>;

void sequentialMultiply(Matrix& left, Matrix& right, Matrix& out, size_t n) {
    agency::experimental::span<size_t> left_data(left.data(), n);
    agency::experimental::span<size_t> right_data(right.data(), n);
    size_t* out_ptr = out.data();

    auto left_rows = agency::experimental::tile_evenly(left_data, n);
    auto right_rows = agency::experimental::tile_evenly(right_data, n);
    agency::bulk_invoke(agency::seq(n), [=](agency::sequenced_agent& outer)
    {
        auto left_row = left_rows[outer.index()];
        agency::bulk_invoke(agency::seq(n), [=](agency::sequenced_agent& inner)
        {
            auto right_row = right_rows[inner.index()];
            for (int k = 0; k < n; ++k) {
                out_ptr[n * outer.index() + inner.index()] += left_row[k] * right_row[k];
            }
        });
    });
}

int main()
{
    size_t n = 1 << 10;

    Matrix a(n*n, 1);
    Matrix b(n*n, 1);
    Matrix c(n*n, 0);

    Matrix reference(n*n, n);
    
    // execute sequentially in the current thread
    clock_t begin_time = clock();
    sequentialMultiply(a, b, c, n);
    clock_t difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Sequential Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);
/*

    // execute in parallel on the CPU
    begin_time = clock();
    bulk_invoke(par(n*n), [=](parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[n*row + col] += a_ptr[n*row + k] *
                                  b_ptr[n*k + col];
        }
    });
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel CPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on a GPU
    begin_time = clock();
    cuda::grid_executor gpu;
    bulk_invoke(par(n*n).on(gpu), [=] __device__ (parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[n*row + col] += a_ptr[n*row + k] *
                                  b_ptr[n*k + col];
        }
    });
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel Single GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on all GPUs in the system
    begin_time = clock();
    cuda::multidevice_executor all_gpus;
    bulk_invoke(par(n).on(all_gpus), [=] __device__ (parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[n*row + col] += a_ptr[n*row + k] *
                                  b_ptr[n*k + col];
        }
    });
    difference = clock() - begin_time;

    assert(c == reference);
    std::fill(c.begin(), c.end(), 0);

    printf("Parallel All GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);
*/
    std::cout << "OK" << std::endl;
    return 0;
}
