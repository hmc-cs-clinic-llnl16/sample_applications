#include "hip/hip_runtime.h"
#include <agency/agency.hpp>
#include <agency/cuda.hpp>
#include <vector>
#include <cassert>
#include <iostream>
#include <ctime>

int main()
{
    using namespace agency;
    // allocate data in GPU memory
    using vector = std::vector<size_t, cuda::managed_allocator<size_t>>;
    using matrix = std::vector<vector, cuda::managed_allocator<vector>>;
    
    size_t n = 1 << 10;
    
    matrix a(n, vector(n, 1));
    matrix b(n, vector(n, 1));
    matrix c(n, vector(n, 0));
    
    vector* a_ptr = a.data();
    vector* b_ptr = b.data();
    vector* c_ptr = c.data();
    
    // execute sequentially in the current thread
    clock_t begin_time = clock();
    bulk_invoke(seq(n*n), [=](sequenced_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[row].data()[col] += a_ptr[row].data()[k] * 
                                      b_ptr[k].data()[col];
        }
    });
    clock_t difference = clock() - begin_time;

    for (auto i = c.begin(); i < c.end(); ++i) {
        for (auto j = i -> begin(); j < i -> end(); ++j) {
            assert(*j == n);
            *j = 0;
        };
    }

    printf("Sequential Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on the CPU
    begin_time = clock();
    bulk_invoke(par(n*n), [=](parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[row].data()[col] += a_ptr[row].data()[k] * 
                                      b_ptr[k].data()[col];
        }
    });
    difference = clock() - begin_time;

    for (auto i = c.begin(); i < c.end(); ++i) {
        for (auto j = i -> begin(); j < i -> end(); ++j) {
            assert(*j == n);
            *j = 0;
        };
    }

    printf("Parallel CPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on a GPU
    begin_time = clock();
    cuda::grid_executor gpu;
    bulk_invoke(par(n*n).on(gpu), [=] __device__ (parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[row].data()[col] += a_ptr[row].data()[k] *
                                      b_ptr[k].data()[col];
        }
    });
    difference = clock() - begin_time;

    for (auto i = c.begin(); i < c.end(); ++i) {
        for (auto j = i -> begin(); j < i -> end(); ++j) {
            assert(*j == n);
            *j = 0;
        };
    }

    printf("Parallel Single GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    // execute in parallel on all GPUs in the system
    begin_time = clock();
    cuda::multidevice_executor all_gpus;
    bulk_invoke(par(n).on(all_gpus), [=] __device__ (parallel_agent& self)
    {
        int row = self.index() / n;
        int col = self.index() % n;

        for (int k = 0; k < n; ++k) {
            c_ptr[row].data()[col] += a_ptr[row].data()[k] *
                                      b_ptr[k].data()[col];
        }
    });
    difference = clock() - begin_time;

    for (auto i = c.begin(); i < c.end(); ++i) {
        for (auto j = i -> begin(); j < i -> end(); ++j) {
            assert(*j == n);
            *j = 0;
        };
    }

    printf("Parallel All GPU Execution took %ld clicks (%f seconds).\n", difference, ((float) difference)/CLOCKS_PER_SEC);

    std::cout << "OK" << std::endl;
    return 0;
}
