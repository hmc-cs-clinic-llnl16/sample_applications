#include "hip/hip_runtime.h"
#include "RAJA/RAJA.hxx"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mmult.cuh"

__global__ void mmultKernel(const double* __restrict__ left, 
                            const double* __restrict__ right, 
                            double* __restrict__ result, 
                            const size_t numRows, const size_t numCols)
{
    const int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int stepSize = blockDim.x * gridDim.x;


    for (int i = threadIndex; i < numRows * numCols; i += stepSize) {
        int row = (i - numCols) / numRows;
        int col = (i - numCols) % numRows;
        
        double tmp = 0.0;
        for (int k = 0; k < numCols; ++k) {
            tmp += left[row * numCols + k] * right[k * numCols + col];
        }
        result[row * numCols + col] = tmp;
    }
}

template <size_t BLOCK_SIZE>
double* mmultGpu(double* left, double* right, size_t numRows, size_t numCols)
{
    double* result;
    hipMallocManaged((void**)&result,
                                 numRows * numCols * sizeof(double),
                                 hipMemAttachGlobal);
    hipMemset(result, 0, numRows * numCols * sizeof(double));
    hipDeviceSynchronize();

    size_t gridSize = RAJA_DIVIDE_CEILING_INT(numRows * numCols, BLOCK_SIZE);
    gridSize = RAJA_MIN(gridSize, RAJA_CUDA_MAX_NUM_BLOCKS);

    mmultKernel<<<gridSize, BLOCK_SIZE>>>(left, right, result, numRows, numCols);
    hipDeviceSynchronize();

    return result;
}

